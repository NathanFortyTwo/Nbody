#include "hip/hip_runtime.h"
/*
** nbody_brute_force.c - nbody simulation using the brute-force algorithm (O(n*n))
** Cuda version
**/

#include <stdio.h>
#include <stdlib.h>
#include <pthread.h>
#include <math.h>
#include <sys/time.h>
#include <assert.h>
#include <unistd.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>

#ifdef DISPLAY
#include <X11/Xlib.h>
#include <X11/Xutil.h>
#endif

#include "ui.h"
#include "nbody.h"
#include "cuda_stuff.cuh"

FILE* f_out = NULL;

int nparticles = 10; /* number of particles */
float T_FINAL = 1.0; /* simulation end time */
particle_t* particles;

double sum_speed_sq = 0;
double max_acc = 0;
double max_speed = 0;

/** CUDA **/
hipError_t err;
particle_t* d_particles;
dim3 threadsPerBlock;
dim3 numBlocks;

void init()
{
  /** Initialisation des variables nbthreadbyblock et nbblockbygrid **/
  threadsPerBlock = dim3(256);
  numBlocks = dim3((nparticles + threadsPerBlock.x - 1) / threadsPerBlock.x);
  printf("Threads per block: %d\n", threadsPerBlock.x);
  printf("Blocks per grid: %d\n", numBlocks.x);
}


#ifdef DISPLAY
extern Display* theDisplay; /* These three variables are required to open the */
extern GC theGC;            /* particle plotting window.  They are externally */
extern Window theMain;      /* declared in ui.h but are also required here.   */
#endif

/* compute the force that a particle with position (x_pos, y_pos) and mass 'mass'
 * applies to particle p
 * Runs as a kernel on the GPU
 */
__device__ __host__ void compute_force(particle_t* p, double x_pos, double y_pos, double mass) {
  double x_sep, y_sep, dist_sq, grav_base;

  x_sep = x_pos - p->x_pos;
  y_sep = y_pos - p->y_pos;
  dist_sq = MAX((x_sep * x_sep) + (y_sep * y_sep), 0.01);

  /* Use the 2-dimensional gravity rule: F = d * (GMm/d^2) */
  grav_base = GRAV_CONSTANT * (p->mass) * (mass) / dist_sq;

  p->x_force += grav_base * x_sep;
  p->y_force += grav_base * y_sep;
}

/* compute all forces that all particles apply to all others
 * Runs as a kernel on the GPU
 */
__global__ void compute_forces_kernel(particle_t* d_particles, int nparticles)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < nparticles)
  {
    int j;
    d_particles[i].x_force = 0;
    d_particles[i].y_force = 0;
    for (j = 0; j < nparticles; j++)
    {
      particle_t* p = &d_particles[j];
      /* compute the force of particle j on particle i */
      compute_force(&d_particles[i], p->x_pos, p->y_pos, p->mass);
    }
  }
}

/* compute the new position/velocity */
void move_particle(particle_t* p, double step) {

  p->x_pos += (p->x_vel) * step;
  p->y_pos += (p->y_vel) * step;
  double x_acc = p->x_force / p->mass;
  double y_acc = p->y_force / p->mass;
  p->x_vel += x_acc * step;
  p->y_vel += y_acc * step;

  /* compute statistics */
  double cur_acc = (x_acc * x_acc + y_acc * y_acc);
  cur_acc = sqrt(cur_acc);
  double speed_sq = (p->x_vel) * (p->x_vel) + (p->y_vel) * (p->y_vel);
  double cur_speed = sqrt(speed_sq);

  sum_speed_sq += speed_sq;
  max_acc = MAX(max_acc, cur_acc);
  max_speed = MAX(max_speed, cur_speed);
}

/*
  Move particles one time step.

  Update positions, velocity, and acceleration.
  Return local computations.
*/
void all_move_particles(double step)
{
  hipMemcpy(d_particles, particles, sizeof(particle_t) * nparticles, hipMemcpyHostToDevice);
  /** Lancement du kernel **/
  compute_forces_kernel << <numBlocks, threadsPerBlock >> > (d_particles, nparticles);
  hipMemcpy(particles, d_particles, sizeof(particle_t) * nparticles, hipMemcpyDeviceToHost);

  /* then move all particles and return statistics */
  int i;
  for (i = 0; i < nparticles; i++)
  {
    move_particle(&particles[i], step);
  }
}


void print_all_particles(FILE* f)
{
  int i;
  for (i = 0; i < nparticles; i++)
  {
    particle_t* p = &particles[i];
    fprintf(f, "particle={pos=(%f,%f), vel=(%f,%f)}\n", p->x_pos, p->y_pos, p->x_vel, p->y_vel);
  }
}

void run_simulation()
{
  double t = 0.0, dt = 0.01;
  while (t < T_FINAL && nparticles > 0)
  {
    /* Update time. */
    t += dt;
    printf("t = %lf\n", t);
    /* Move particles with the current and compute rms velocity. */
    all_move_particles(dt);

    /* Adjust dt based on maximum speed and acceleration--this
       simple rule tries to insure that no velocity will change
       by more than 10% */

    dt = 0.1 * (max_speed) / (max_acc);

  }
}

/*
  Place particles in their initial positions.
*/
void all_init_particles(int num_particles, particle_t* particles)
{
  int    i;
  double total_particle = num_particles;

  for (i = 0; i < num_particles; i++) {
    particle_t* particle = &particles[i];

    particle->x_pos = i * 2.0 / nparticles - 1.0;
    particle->y_pos = 0.0;
    particle->x_vel = 0.0;
    particle->y_vel = particle->x_pos;

    particle->mass = 1.0 + (num_particles + i) / total_particle;
    particle->node = NULL;

  }
}

/*
  Simulate the movement of nparticles particles.
*/
int main(int argc, char** argv)
{
  if (argc >= 2)
  {
    nparticles = atoi(argv[1]);
  }
  if (argc == 3)
  {
    T_FINAL = atof(argv[2]);
  }

  init();

  /* Allocate global shared arrays for the particles data set. */
  /** Allocation memoire sur le host(CPU) **/
  particles = (particle_t*)malloc(sizeof(particle_t) * nparticles);
  all_init_particles(nparticles, particles);

  /** Allocation memoire sur le device(GPU) **/
  err = hipMalloc((void**)&d_particles, sizeof(particle_t) * nparticles);
  gpuErrchk(err);

  /** Transfert mémoire du host vers le device **/
  err = hipMemcpy(d_particles, particles, sizeof(particle_t) * nparticles, hipMemcpyHostToDevice);
  gpuErrchk(err);

  /* Initialize thread data structures */
#ifdef DISPLAY
  /* Open an X window to display the particles */
  simple_init(100, 100, DISPLAY_SIZE, DISPLAY_SIZE);
#endif

  struct timeval t1, t2;
  gettimeofday(&t1, NULL);

  run_simulation();

  gettimeofday(&t2, NULL);

  double duration = (t2.tv_sec - t1.tv_sec) + ((t2.tv_usec - t1.tv_usec) / 1e6);


#ifdef DUMP_RESULT
  FILE* f_out = fopen("particles.log", "w");
  assert(f_out);
  print_all_particles(f_out);
  fclose(f_out);
#endif

  printf("-----------------------------\n");
  printf("nparticles: %d\n", nparticles);
  printf("T_FINAL: %f\n", T_FINAL);
  printf("-----------------------------\n");
  printf("Simulation took %lf s to complete\n", duration);

#ifdef DISPLAY
  clear_display();
  draw_all_particles();
  flush_display();

  printf("Hit return to close the window.");

  getchar();
  /* Close the X window used to display the particles */
  XCloseDisplay(theDisplay);
#endif
  /** Libération de la mémoire **/
  err = hipFree(d_particles);
  gpuErrchk(err);
  free(particles);

  return 0;
}
